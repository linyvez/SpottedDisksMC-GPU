#include "hip/hip_runtime.h"

#include "circle_move.cuh"
#include "hiprand/hiprand_kernel.h"
#include <hip/hip_runtime_api.h>

#include <cmath>
#include <cstdio>
#include <patch.h>
#include <circle_config.h>

#include "periodic_boundary.h"
#include "shared_utilities.h"
#include <random>
#include <algorithm>

__device__
inline int cell_index(double coord) {
    int raw = static_cast<int>(coord / CELL_SIZE);

    if (raw >= Mx) {
        raw -= 1;
    }
    return raw;
}

__device__ static
void d_periodic_boundary(double *dx, double *dy) {
    if ((dc_bc & PERIODIC_X) != 0) {
        *dx = fmod(*dx + 0.5 * Lx, Lx);
        if (*dx < 0) *dx += Lx;
        *dx -= 0.5 * Lx;
    }
    if ((dc_bc & PERIODIC_Y) != 0) {
        *dy = fmod(*dy + 0.5 * Ly, Ly);
        if (*dy < 0) *dy += Ly;
        *dy -= 0.5 * Ly;
    }
}

__device__ static CircleParticle adjust_circle_for_periodic(const CircleParticle ref, const CircleParticle cp) {
    CircleParticle sp_adj = cp;
    double dx = cp.x - ref.x;
    double dy = cp.y - ref.y;

    d_periodic_boundary(&dx, &dy);

    sp_adj.x = ref.x + dx;
    sp_adj.y = ref.y + dy;

    return sp_adj;
}


__device__ static int check_circles_overlap(CircleParticle a, CircleParticle b) {
    CircleParticle b_adj = adjust_circle_for_periodic(a, b);

    double x_diff = a.x - b_adj.x;
    double y_diff = a.y - b_adj.y;

    double distance = x_diff * x_diff + y_diff * y_diff;

    if (distance < PARTICLE_SIZE * PARTICLE_SIZE) {
        return 1;
    }
    return 0;
}


__device__ static int d_is_overlapping_circle(const CircleParticle cp, CircleParticle * circles, LinkedCell cell_struct, int Idx) {
    int cell_ix = cell_index(cp.x);
    int cell_iy = cell_index(cp.y);

    for (int ox = -1; ox <= 1; ox++) {
        for (int oy = -1; oy <= 1; oy++) {
            int ghost_ix = (cell_ix + ox + Mx) % Mx;
            int ghost_iy = (cell_iy + oy + My) % My;
            int ghost_cell = ghost_ix + ghost_iy * Mx;

            int cnt = cell_struct.count[ghost_cell];
            for (int i = 0; i < cnt; i++) {
                int nidx = cell_struct.neighbors[ghost_cell][i];

                if (Idx == nidx) {
                    continue;
                }
                if (check_circles_overlap(circles[nidx], cp)) {
                    return 1;
                }
            }
        }
    }
    return 0;
}


__device__ static double compute_patch_interaction(
    const CircleParticle &a,
    const CircleParticle &b)
{
    CircleParticle b_adj = adjust_circle_for_periodic(a, b);

    double thetaA = 2.0 * atan2(a.q[0], a.q[1]);
    double thetaB = 2.0 * atan2(b_adj.q[0], b_adj.q[1]);
    double cA = cos(thetaA), sA = sin(thetaA);
    double cB = cos(thetaB), sB = sin(thetaB);

    double energy = 0.0;
    constexpr double cutoff2 =  PARTICLE_SIZE * PARTICLE_SIZE;

    for (int i = 0; i < dc_num_patches; ++i) {
        double rx1 = dc_patch[i][0], ry1 = dc_patch[i][1];
        double xi  = a.x + (rx1 * cA - ry1 * sA);
        double yi  = a.y + (rx1 * sA + ry1 * cA);

        for (int j = 0; j < dc_num_patches; ++j) {
            double rx2 = dc_patch[j][0], ry2 = dc_patch[j][1];
            double xj  = b_adj.x + (rx2 * cB - ry2 * sB);
            double yj  = b_adj.y + (rx2 * sB + ry2 * cB);

            double dx = xi - xj;
            double dy = yi - yj;
            d_periodic_boundary(&dx, &dy);

            if (dx*dx + dy*dy < cutoff2) {
                energy += PATCH_STRENGTH;
            }
        }
    }

    return energy;
}



__device__ static double compute_patch_energy(
    const CircleParticle &cp,
    const CircleParticle *circles,
    LinkedCell       cell_struct,
    int              circleIdx)
{
    double sum = 0.0;

    int ix = cell_index(cp.x);
    int iy = cell_index(cp.y);

    for (int dx = -1; dx <= 1; ++dx) {
        for (int dy = -1; dy <= 1; ++dy) {
            int nix = (ix + dx + Mx) % Mx;
            int niy = (iy + dy + My) % My;
            int cell = nix + niy * Mx;

            int cnt = cell_struct.count[cell];
            for (int k = 0; k < cnt; ++k) {
                int j = cell_struct.neighbors[cell][k];
                if (j == circleIdx) continue;
                sum += compute_patch_interaction(cp, circles[j]);
            }
        }
    }

    return sum;
}





__device__ static void insert_circle_in_cells(int circleIndex,
                                              const CircleParticle &cp,
                                              LinkedCell cell_struct) {
    int ix = cell_index(cp.x);
    int iy = cell_index(cp.y);
    int cell = ix + iy * Mx;

    int old_cnt = atomicAdd(&cell_struct.count[cell], 1);

    cell_struct.neighbors[cell][old_cnt] = circleIndex;
}


__device__ static void remove_circle_from_cells(int circleIndex,
                                                const CircleParticle &cp,
                                                LinkedCell cell_struct) {
    int ix = cell_index(cp.x);
    int iy = cell_index(cp.y);
    int cell = ix + iy * Mx;

    int cnt = cell_struct.count[cell];
    int pos = -1;
    for (int s = 0; s < cnt; ++s) {
        if (cell_struct.neighbors[cell][s] == circleIndex) {
            pos = s;
            break;
        }
    }
    if (pos < 0) return;

    int old_cnt = atomicSub(&cell_struct.count[cell], 1);
    int last = old_cnt - 1;

    if (pos != last) {
        atomicExch(&cell_struct.neighbors[cell][pos],
                   cell_struct.neighbors[cell][last]);
    }
}


__device__ static double apply_boundary(double coord, double L) {
    if (dc_bc) {
        if (coord < 0)
            coord += L;
        else if (coord > L)
            coord -= L;
    } else {
        if (coord < 0)
            coord = 0;
        else if (coord > L)
            coord = L;
    }
    return coord;
}

__device__ static void rotate_circle(CircleParticle *sp, hiprandState *rng) {
    double current_angle = 2.0 * atan2(sp->q[0], sp->q[1]);
    float u = hiprand_uniform(rng);

    double range = dc_lflag != 0 && u < LRMOVE ? LRROTMAX : RROTMAX;

    float v = hiprand_uniform(rng);

    double angle = v * 2 * range - range;


    current_angle += angle;
    current_angle = fmod(current_angle, 2 * M_PI);
    if (current_angle < 0)
        current_angle += 2 * M_PI;

    sp->q[0] = sin(current_angle / 2);
    sp->q[1] = cos(current_angle / 2);
}



__device__ static void move_circle(CircleParticle *cp, hiprandState *rng) {
    double range = dc_lflag != 0 && (hiprand_uniform(rng) < LRMOVE) ? LRDISPMAX : RDISPMAX;

    double dx = (hiprand_uniform(rng) * 2.0 - 1.0) * range;
    double dy = (hiprand_uniform(rng) * 2.0 - 1.0) * range;

    cp->x += dx;
    cp->y += dy;

    cp->x = apply_boundary(cp->x, Lx);
    cp->y = apply_boundary(cp->y, Ly);
}


static void shuffle_c(int *array) {
    static std::random_device rd;
    static std::mt19937 gen(rd());

    std::shuffle(array, array + 4, gen);
}


__global__ static void initRNG_c(hiprandState *states, unsigned long seed) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(
        seed,
        tid,
        0,
        &states[tid]
    );
}



__global__ static void mc_step_kernel_c(int color,
                               hiprandState *rngStates,
                               CircleParticle *circles,
                               LinkedCell cell_struct) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned cellX = tid % Mx;
    unsigned cellY = (tid / Mx) % My;
    unsigned myColor = (cellX & 1) << 1 | (cellY & 1);


    if (tid >= NUM_CELLS || myColor != color) {
        return;
    }

    hiprandState localState = rngStates[tid];


    int cnt = cell_struct.count[tid];
    if (cnt == 0) return;

    int r = static_cast<int>(hiprand_uniform_double(&localState) * cnt);
    if (r == cnt) r = cnt - 1;

    int circleIdx = cell_struct.neighbors[tid][r];


    CircleParticle candidate = circles[circleIdx];

    float rm = hiprand_uniform(&localState);

    if (rm < RMOVE) {
        move_circle(&candidate, &localState);
    } else {
        rotate_circle(&candidate, &localState);
    }

    if (!d_is_overlapping_circle(candidate, circles, cell_struct, circleIdx)) {
        double myDE = compute_patch_energy(candidate, circles, cell_struct, circleIdx) - compute_patch_energy(
                          circles[circleIdx], circles, cell_struct, circleIdx);


        if (myDE <= 0 || exp(-myDE / KT) > hiprand_uniform(&localState)) {
            int new_cell = cell_index(candidate.x) + cell_index(candidate.y) * Mx;
            if (new_cell != tid) {
                remove_circle_from_cells(circleIdx, circles[circleIdx], cell_struct);
                insert_circle_in_cells(circleIdx, candidate, cell_struct);
            }

            circles[circleIdx] = candidate;
        }
    }
    rngStates[tid] = localState;
}

__constant__ int dc_num_patches;
__constant__ BoundaryCondition dc_bc;
__constant__ int dc_lflag;
__constant__ double dc_patch[MAX_PATCHES][2];


void animate_movement_c(int totalCircles, const CircleParticle *circles, LinkedCell cell_struct, int lflag) {
    hipMemcpyToSymbol(HIP_SYMBOL(dc_bc), &h_bc, sizeof(h_bc));
    hipMemcpyToSymbol(HIP_SYMBOL(dc_num_patches), &h_num_patches, sizeof(h_num_patches));
    hipMemcpyToSymbol(HIP_SYMBOL(dc_lflag), &lflag, sizeof(lflag));
    hipMemcpyToSymbol(HIP_SYMBOL(dc_patch), h_patch, h_num_patches * 2 * sizeof(double));


    CircleParticle *d_circles;

    CircleParticle *d_snapshots = nullptr;
    size_t snapshotCount = static_cast<size_t>(ANIMATION_STEPS) * totalCircles;
    hipMalloc(&d_snapshots, snapshotCount * sizeof(CircleParticle));

    hipMalloc(&d_circles, totalCircles * sizeof(CircleParticle));
    hipMemcpy(d_circles, circles, totalCircles * sizeof(CircleParticle), hipMemcpyHostToDevice);

    int *d_count_mem;
    int *d_neigh_mem;
    LinkedCell d_cell_struct;


    hipMalloc(&d_count_mem, sizeof(int) * NUM_CELLS);
    hipMalloc(&d_neigh_mem, sizeof(int) * NUM_CELLS * MAX_NEIGH);

    hipMemcpy(d_count_mem,
               cell_struct.count,
               sizeof(int) * NUM_CELLS,
               hipMemcpyHostToDevice);

    hipMemcpy(
        d_neigh_mem,
        &cell_struct.neighbors[0][0],
        sizeof(int) * NUM_CELLS * MAX_NEIGH,
        hipMemcpyHostToDevice
    );

    d_cell_struct.count = d_count_mem;
    d_cell_struct.neighbors = reinterpret_cast<int (*)[MAX_NEIGH]>(d_neigh_mem);


    int threads = 32;
    int blocks = (NUM_CELLS + threads - 1) / threads;

    hiprandState *d_rngStates;
    size_t nThreads = blocks * threads;
    hipMalloc(&d_rngStates, nThreads * sizeof(hiprandState));


    int colors[4] = {0, 1, 2, 3};


    initRNG_c<<<blocks, threads>>>(d_rngStates, 1234UL);
    hipDeviceSynchronize();

    const clock_t start = clock();

    for (int step = 0; step < ANIMATION_STEPS; step++) {
        shuffle_c(colors);

        for (int color: colors) {
            mc_step_kernel_c<<<blocks,threads>>>(
                color,
                d_rngStates,
                d_circles,
                d_cell_struct
            );
        }
        CircleParticle *dst = d_snapshots + static_cast<size_t>(step) * totalCircles;

        hipMemcpy(dst,
                   d_circles,
                   totalCircles * sizeof(CircleParticle),
                   hipMemcpyDeviceToDevice);
    }

    const clock_t end = clock();

    double elapsed_ms = (double)(end - start) * 1000.0 / (double)CLOCKS_PER_SEC;
    printf("Elapsed time: %.3f ms\n", elapsed_ms);

    CircleParticle *h_snapshots = nullptr;
    hipHostMalloc(&h_snapshots, snapshotCount * sizeof(CircleParticle));

    hipMemcpy(h_snapshots,
               d_snapshots,
               snapshotCount * sizeof(CircleParticle),
               hipMemcpyDeviceToHost);


    FILE *f = fopen("data/circle_animation.xyz", "w");
    if (!f) {
        printf("Error opening animation file\n");
        return;
    }

    for (int step = 0; step < ANIMATION_STEPS; ++step) {
        CircleParticle *frame = h_snapshots + static_cast<size_t>(step) * totalCircles;
        write_file_c(f, frame, totalCircles);
    }


    hipFree(d_circles);
    hipFree(d_count_mem);
    hipFree(d_neigh_mem);
    hipFree(d_rngStates);
    hipHostFree(h_snapshots);
    hipFree(d_snapshots);

    fclose(f);
}
