#include "hip/hip_runtime.h"
#include "square_move.cuh"
#include "hiprand/hiprand_kernel.h"
#include <hip/hip_runtime_api.h>

#include <cmath>
#include <cstdio>
#include <patch.h>
#include <square_config.h>

#include "periodic_boundary.h"
#include "shared_utilities.h"
#include <random>
#include <algorithm>

__device__
inline int cell_index(double coord) {
    int raw = static_cast<int>(coord / CELL_SIZE);

    if (raw >= Mx) {
        raw -= 1;
    }
    return raw;
}

__device__ static
void d_periodic_boundary(double *dx, double *dy) {
    if ((d_bc & PERIODIC_X) != 0) {
        *dx = fmod(*dx + 0.5 * Lx, Lx);
        if (*dx < 0) *dx += Lx;
        *dx -= 0.5 * Lx;
    }
    if ((d_bc & PERIODIC_Y) != 0) {
        *dy = fmod(*dy + 0.5 * Ly, Ly);
        if (*dy < 0) *dy += Ly;
        *dy -= 0.5 * Ly;
    }
}

__device__ static SquareParticle d_adjust_square_for_periodic(const SquareParticle &ref, const SquareParticle &sp) {
    SquareParticle sp_adj = sp;
    double dx = sp.x - ref.x;
    double dy = sp.y - ref.y;

    d_periodic_boundary(&dx, &dy);

    sp_adj.x = ref.x + dx;
    sp_adj.y = ref.y + dy;

    return sp_adj;
}

__device__ static void d_normalize(double *ax, double *ay) {
    const double len = sqrt((*ax) * (*ax) + (*ay) * (*ay));
    if (len > 1e-10) {
        *ax /= len;
        *ay /= len;
    }
}

__device__ static void d_project_polygon(const double corners[][2],
                                         double ax, double ay,
                                         double *minProj, double *maxProj) {
    constexpr int num = 4;
    double dot = corners[0][0] * ax + corners[0][1] * ay;
    *minProj = *maxProj = dot;
    for (int i = 1; i < num; i++) {
        dot = corners[i][0] * ax + corners[i][1] * ay;
        if (dot < *minProj)
            *minProj = dot;
        if (dot > *maxProj)
            *maxProj = dot;
    }
}

__device__ static int d_intervals_overlap(double Amin, double Amax,
                                          double Bmin, double Bmax) {
    if (Amax < Bmin)
        return 0;
    if (Bmax < Amin)
        return 0;
    return 1;
}

__device__ static void d_compute_square_corners(const SquareParticle &sp, double corners[4][2]) {
    double half = PARTICLE_SIZE / 2.0;
    double angle = 2.0 * atan2(sp.q[0], sp.q[1]);
    double cosA = cos(angle);
    double sinA = sin(angle);

    double local[4][2] = {
        {-half, half},
        {-half, -half},
        {half, -half},
        {half, half}
    };
    for (int i = 0; i < 4; i++) {
        corners[i][0] = sp.x + local[i][0] * cosA - local[i][1] * sinA;
        corners[i][1] = sp.y + local[i][0] * sinA + local[i][1] * cosA;
    }
}

__device__ static void d_get_square_axes(const double corners[4][2], double axes[2][2]) {
    double e0x = corners[1][0] - corners[0][0];
    double e0y = corners[1][1] - corners[0][1];
    axes[0][0] = -e0y;
    axes[0][1] = e0x;
    d_normalize(&axes[0][0], &axes[0][1]);

    double e1x = corners[2][0] - corners[1][0];
    double e1y = corners[2][1] - corners[1][1];
    axes[1][0] = -e1y;
    axes[1][1] = e1x;
    d_normalize(&axes[1][0], &axes[1][1]);
}

__device__ static int check_squares_overlap(const SquareParticle &a, const SquareParticle &b) {
    SquareParticle b_adj = d_adjust_square_for_periodic(a, b);
    double A[4][2], B[4][2];
    d_compute_square_corners(a, A);
    d_compute_square_corners(b_adj, B);

    double axesA[2][2];
    double axesB[2][2];
    d_get_square_axes(A, axesA);
    d_get_square_axes(B, axesB);

#define TEST_AXIS(ax, ay)                                \
     do                                                   \
     {                                                    \
         double Amin, Amax, Bmin, Bmax;                   \
         d_project_polygon(A, (ax), (ay), &Amin, &Amax); \
         d_project_polygon(B, (ax), (ay), &Bmin, &Bmax); \
         if (!d_intervals_overlap(Amin, Amax, Bmin, Bmax))  \
         {                                                \
             return 0;                                    \
         }                                                \
     } while (0)

    TEST_AXIS(axesA[0][0], axesA[0][1]);
    TEST_AXIS(axesA[1][0], axesA[1][1]);
    TEST_AXIS(axesB[0][0], axesB[0][1]);
    TEST_AXIS(axesB[1][0], axesB[1][1]);

    return 1;
}


__device__ int static d_is_overlapping_square(const SquareParticle &sp, const SquareParticle *squares,
                                              LinkedCell cell_struct, int squareIdx) {
    int cell_ix = cell_index(sp.x);
    int cell_iy = cell_index(sp.y);

    for (int ox = -1; ox <= 1; ox++) {
        for (int oy = -1; oy <= 1; oy++) {
            int ghost_ix = (cell_ix + ox + Mx) % Mx;
            int ghost_iy = (cell_iy + oy + My) % My;
            int ghost_cell = ghost_ix + ghost_iy * Mx;

            int cnt = cell_struct.count[ghost_cell];
            for (int i = 0; i < cnt; i++) {
                int nidx = cell_struct.neighbors[ghost_cell][i];

                if (nidx == squareIdx) {
                    continue;
                }

                if (check_squares_overlap(squares[nidx], sp)) {
                    return 1;
                }
            }
        }
    }
    return 0;
}


__device__ double compute_patch_interaction(
    const SquareParticle &sp1,
    const SquareParticle &sp2) {
    SquareParticle sp2_adj = d_adjust_square_for_periodic(sp1, sp2);

    double angle1 = 2.0 * atan2(sp1.q[0], sp1.q[1]);
    double cos1 = cos(angle1), sin1 = sin(angle1);
    double angle2 = 2.0 * atan2(sp2_adj.q[0], sp2_adj.q[1]);
    double cos2 = cos(angle2), sin2 = sin(angle2);

    int n = d_num_patches;
    double gx1[MAX_PATCHES], gy1[MAX_PATCHES];
    double gx2[MAX_PATCHES], gy2[MAX_PATCHES];

    for (int i = 0; i < n; ++i) {
        double rx = d_patch[i][0], ry = d_patch[i][1];
        gx1[i] = sp1.x + rx * cos1 - ry * sin1;
        gy1[i] = sp1.y + rx * sin1 + ry * cos1;
        gx2[i] = sp2_adj.x + rx * cos2 - ry * sin2;
        gy2[i] = sp2_adj.y + rx * sin2 + ry * cos2;
    }

    double energy = 0.0;
    constexpr double R2 = (2.0 * PATCH_RADIUS) * (2.0 * PATCH_RADIUS);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            double dx = gx1[i] - gx2[j];
            double dy = gy1[i] - gy2[j];
            d_periodic_boundary(&dx, &dy);
            double dist2 = dx * dx + dy * dy;
            if (dist2 < R2) {
                energy += PATCH_STRENGTH;
            }
        }
    }

    return energy;
}


__device__ static double compute_patch_energy(const SquareParticle &sp, const SquareParticle *squares,
                                              LinkedCell cell_struct, int squareIdx) {
    double sum = 0.0;

    int cell_ix = cell_index(sp.x);
    int cell_iy = cell_index(sp.y);

    for (int ox = -1; ox <= 1; ox++) {
        for (int oy = -1; oy <= 1; oy++) {
            int ghost_ix = (cell_ix + ox + Mx) % Mx;
            int ghost_iy = (cell_iy + oy + My) % My;
            int ghost_cell = ghost_ix + ghost_iy * Mx;

            int cnt = cell_struct.count[ghost_cell];
            for (int i = 0; i < cnt; i++) {
                int nidx = cell_struct.neighbors[ghost_cell][i];

                if (nidx == squareIdx) {
                    continue;
                }

                sum += compute_patch_interaction(squares[nidx], sp);
            }
        }
    }
    return sum;
}




__device__ static void insert_square_in_cells(int squareIndex,
                                              const SquareParticle &sp,
                                              LinkedCell cell_struct) {
    int ix = cell_index(sp.x);
    int iy = cell_index(sp.y);
    int cell = ix + iy * Mx;

    int old_cnt = atomicAdd(&cell_struct.count[cell], 1);

    cell_struct.neighbors[cell][old_cnt] = squareIndex;
}


__device__ static void remove_square_from_cells(int squareIndex,
                                                const SquareParticle &sp,
                                                LinkedCell cell_struct) {
    int ix = cell_index(sp.x);
    int iy = cell_index(sp.y);
    int cell = ix + iy * Mx;

    int cnt = cell_struct.count[cell];
    int pos = -1;
    for (int s = 0; s < cnt; ++s) {
        if (cell_struct.neighbors[cell][s] == squareIndex) {
            pos = s;
            break;
        }
    }
    if (pos < 0) return;

    int old_cnt = atomicSub(&cell_struct.count[cell], 1);
    int last = old_cnt - 1;

    if (pos != last) {
        atomicExch(&cell_struct.neighbors[cell][pos],
                   cell_struct.neighbors[cell][last]);
    }
}


__device__ double apply_boundary(double coord, double L) {
    if (d_bc) {
        if (coord < 0)
            coord += L;
        else if (coord > L)
            coord -= L;
    } else {
        if (coord < 0)
            coord = 0;
        else if (coord > L)
            coord = L;
    }
    return coord;
}

__device__ static void rotate_square(SquareParticle *sp, hiprandState *rng) {
    double current_angle = 2.0 * atan2(sp->q[0], sp->q[1]);
    float u = hiprand_uniform(rng);

    double range = d_lflag != 0 && u < LRMOVE ? LRROTMAX : RROTMAX;

    float v = hiprand_uniform(rng);

    double angle = v * 2 * range - range;


    current_angle += angle;
    current_angle = fmod(current_angle, 2 * M_PI);
    if (current_angle < 0)
        current_angle += 2 * M_PI;

    sp->q[0] = sin(current_angle / 2);
    sp->q[1] = cos(current_angle / 2);
}

__device__ static void move_square(SquareParticle *sp, hiprandState *rng) {
    double range = d_lflag != 0 && (hiprand_uniform(rng) < LRMOVE) ? LRDISPMAX : RDISPMAX;

    double dx = (hiprand_uniform(rng) * 2.0 - 1.0) * range;
    double dy = (hiprand_uniform(rng) * 2.0 - 1.0) * range;

    sp->x += dx;
    sp->y += dy;

    sp->x = apply_boundary(sp->x, Lx);
    sp->y = apply_boundary(sp->y, Ly);
}


static void shuffle(int *array) {
    static std::random_device rd;
    static std::mt19937 gen(rd());

    std::shuffle(array, array + 4, gen);
}


__global__ void initRNG(hiprandState *states, unsigned long seed) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(
        seed,
        tid,
        0,
        &states[tid]
    );
}



__global__ void mc_step_kernel(int color,
                               hiprandState *rngStates,
                               SquareParticle *squares,
                               LinkedCell cell_struct) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned cellX = tid % Mx;
    unsigned cellY = (tid / Mx) % My;
    unsigned myColor = (cellX & 1) << 1 | (cellY & 1);


    if (tid >= NUM_CELLS || myColor != color) {
        return;
    }

    hiprandState localState = rngStates[tid];


    int cnt = cell_struct.count[tid];
    if (cnt == 0) return;

    int r = static_cast<int>(hiprand_uniform_double(&localState) * cnt);
    if (r == cnt) r = cnt - 1;

    int squareIdx = cell_struct.neighbors[tid][r];


    SquareParticle candidate = squares[squareIdx];

    float rm = hiprand_uniform(&localState);

    if (rm < RMOVE) {
        move_square(&candidate, &localState);
    } else {
        rotate_square(&candidate, &localState);
    }

    if (!d_is_overlapping_square(candidate, squares, cell_struct, squareIdx)) {
        double myDE = compute_patch_energy(candidate, squares, cell_struct, squareIdx) - compute_patch_energy(
                          squares[squareIdx], squares, cell_struct, squareIdx);


        if (myDE <= 0 || exp(-myDE / KT) > hiprand_uniform(&localState)) {
            int new_cell = cell_index(candidate.x) + cell_index(candidate.y) * Mx;
            if (new_cell != tid) {
                remove_square_from_cells(squareIdx, squares[squareIdx], cell_struct);
                insert_square_in_cells(squareIdx, candidate, cell_struct);
            }

            squares[squareIdx] = candidate;
        }
    }
    rngStates[tid] = localState;
}

__constant__ int d_num_patches;
__constant__ BoundaryCondition d_bc;
__constant__ int d_lflag;
__constant__ double d_patch[MAX_PATCHES][2];


void animate_movement(const int totalSquares, const SquareParticle *squares, LinkedCell cell_struct, int lflag) {
    hipMemcpyToSymbol(HIP_SYMBOL(d_bc), &h_bc, sizeof(h_bc));
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_patches), &h_num_patches, sizeof(h_num_patches));
    hipMemcpyToSymbol(HIP_SYMBOL(d_lflag), &lflag, sizeof(lflag));
    hipMemcpyToSymbol(HIP_SYMBOL(d_patch), h_patch, h_num_patches * 2 * sizeof(double));


    SquareParticle *d_squares;

    SquareParticle *d_snapshots = nullptr;
    size_t snapshotCount = static_cast<size_t>(ANIMATION_STEPS) * totalSquares;
    hipMalloc(&d_snapshots, snapshotCount * sizeof(SquareParticle));

    hipMalloc(&d_squares, totalSquares * sizeof(SquareParticle));
    hipMemcpy(d_squares, squares, totalSquares * sizeof(SquareParticle), hipMemcpyHostToDevice);

    int *d_count_mem;
    int *d_neigh_mem;
    LinkedCell d_cell_struct;


    hipMalloc(&d_count_mem, sizeof(int) * NUM_CELLS);
    hipMalloc(&d_neigh_mem, sizeof(int) * NUM_CELLS * MAX_NEIGH);

    hipMemcpy(d_count_mem,
               cell_struct.count,
               sizeof(int) * NUM_CELLS,
               hipMemcpyHostToDevice);

    hipMemcpy(
        d_neigh_mem,
        &cell_struct.neighbors[0][0],
        sizeof(int) * NUM_CELLS * MAX_NEIGH,
        hipMemcpyHostToDevice
    );

    d_cell_struct.count = d_count_mem;
    d_cell_struct.neighbors = reinterpret_cast<int (*)[MAX_NEIGH]>(d_neigh_mem);


    int threads = 32;
    int blocks = (NUM_CELLS + threads - 1) / threads;

    hiprandState *d_rngStates;
    size_t nThreads = blocks * threads;
    hipMalloc(&d_rngStates, nThreads * sizeof(hiprandState));


    int colors[4] = {0, 1, 2, 3};


    initRNG<<<blocks, threads>>>(d_rngStates, 1234UL);
    hipDeviceSynchronize();

    const clock_t start = clock();

    for (int step = 0; step < ANIMATION_STEPS; step++) {
        shuffle(colors);

        for (int color: colors) {
            mc_step_kernel<<<blocks,threads>>>(
                color,
                d_rngStates,
                d_squares,
                d_cell_struct
            );
        }
        SquareParticle *dst = d_snapshots + static_cast<size_t>(step) * totalSquares;

        hipMemcpy(dst,
                   d_squares,
                   totalSquares * sizeof(SquareParticle),
                   hipMemcpyDeviceToDevice);
    }

    const clock_t end = clock();

    double elapsed_ms = (double)(end - start) * 1000.0 / (double)CLOCKS_PER_SEC;
    printf("Elapsed time: %.3f ms\n", elapsed_ms);

    SquareParticle *h_snapshots = nullptr;
    hipHostMalloc(&h_snapshots, snapshotCount * sizeof(SquareParticle));

    hipMemcpy(h_snapshots,
               d_snapshots,
               snapshotCount * sizeof(SquareParticle),
               hipMemcpyDeviceToHost);


    FILE *f = fopen("data/square_animation.xyz", "w");
    if (!f) {
        printf("Error opening animation file\n");
        return;
    }

    for (int step = 0; step < ANIMATION_STEPS; ++step) {
        SquareParticle *frame = h_snapshots + static_cast<size_t>(step) * totalSquares;
        write_file(f, frame, totalSquares);
    }


    hipFree(d_squares);
    hipFree(d_count_mem);
    hipFree(d_neigh_mem);
    hipFree(d_rngStates);
    hipHostFree(h_snapshots);
    hipFree(d_snapshots);

    fclose(f);
}
